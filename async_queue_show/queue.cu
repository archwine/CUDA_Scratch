#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned" << 
            static_cast<int>(error) << ":"  <<  
            hipGetErrorString(error) << std::endl; 
        return 1;
    }

    for (int device = 0; device < deviceCount; ++ device) {
        hipDeviceProp_t deviceProp;
        error =  hipGetDeviceProperties(&deviceProp, device);

        if (error != hipSuccess) {
            std::cerr  << "hipGetDeviceProperties returned" <<
                static_cast<int>(error) << ":"  <<
                hipGetErrorString(error) << std::endl;
                return 1;
        }

        std::cout << "Device"  << device << ":"  <<  
            deviceProp.name << std::endl;
        std::cout <<  "asynEngineCount: " << 
            deviceProp.asyncEngineCount << std::endl;
    }
}