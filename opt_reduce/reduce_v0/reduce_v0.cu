#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

__global__ void reduce0(float *d_input, float *d_output){
    __shared__ float s_data[THREAD_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;

    s_data[tid] = d_input[i];

    __syncthreads();

    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) d_output[blockIdx.x] = s_data[0];
}


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}

int main(){
    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));

    int block_num=N/THREAD_PER_BLOCK;
    float *out=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));
    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid( N / THREAD_PER_BLOCK,1);
    dim3 Block( THREAD_PER_BLOCK,1);

    reduce0<<<Grid,Block>>>(d_a,d_out);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);

    if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
}